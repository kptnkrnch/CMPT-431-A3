#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <cmath>
#include "hist-equ.cuh"



PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int hist[256];
    
    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
	
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);
    return result;
}

/*
//not called anywhere nor does it seem we even want to use this. keep just in case
PPM_IMG contrast_enhancement_c_rgb(PPM_IMG img_in)
{
    PPM_IMG result;
    int hist[256];
    
    result.w = img_in.w;
    result.h = img_in.h;
    result.img_r = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_g = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_b = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    histogram(hist, img_in.img_r, img_in.h * img_in.w, 256);
    histogram_equalization(result.img_r,img_in.img_r,hist,result.w*result.h, 256);
    histogram(hist, img_in.img_g, img_in.h * img_in.w, 256);
    histogram_equalization(result.img_g,img_in.img_g,hist,result.w*result.h, 256);
    histogram(hist, img_in.img_b, img_in.h * img_in.w, 256);
    histogram_equalization(result.img_b,img_in.img_b,hist,result.w*result.h, 256);

    return result;
}
*/


PPM_IMG contrast_enhancement_c_yuv(PPM_IMG img_in)
{
    YUV_IMG yuv_med;
    PPM_IMG result;
    
    unsigned char * y_equ;
    int hist[256];
    
    yuv_med = rgb2yuv(img_in);
    y_equ = (unsigned char *)malloc(yuv_med.h*yuv_med.w*sizeof(unsigned char));
    
    histogram(hist, yuv_med.img_y, yuv_med.h * yuv_med.w, 256);
	
    histogram_equalization(y_equ,yuv_med.img_y,hist,yuv_med.h * yuv_med.w, 256);

    free(yuv_med.img_y);
    yuv_med.img_y = y_equ;
    
    result = yuv2rgb(yuv_med);
    free(yuv_med.img_y);
    free(yuv_med.img_u);
    free(yuv_med.img_v);
    
    return result;
}

PPM_IMG contrast_enhancement_c_hsl(PPM_IMG img_in)
{
    HSL_IMG hsl_med;
    PPM_IMG result;
    
    unsigned char * l_equ;
    int hist[256];

    hsl_med = rgb2hsl(img_in);
    l_equ = (unsigned char *)malloc(hsl_med.height*hsl_med.width*sizeof(unsigned char));

    histogram(hist, hsl_med.l, hsl_med.height * hsl_med.width, 256);
    histogram_equalization(l_equ, hsl_med.l,hist,hsl_med.width*hsl_med.height, 256);
    
    free(hsl_med.l);
    hsl_med.l = l_equ;

    result = hsl2rgb(hsl_med);
    free(hsl_med.h);
    free(hsl_med.s);
    free(hsl_med.l);
    return result;
}


//Convert RGB to HSL, assume R,G,B in [0, 255]
//Output H, S in [0.0, 1.0] and L in [0, 255]
HSL_IMG rgb2hsl(PPM_IMG img_in)
{
    int i;
    float H, S, L;
    HSL_IMG img_out;// = (HSL_IMG *)malloc(sizeof(HSL_IMG));
    img_out.width  = img_in.w;
    img_out.height = img_in.h;
    img_out.h = (float *)malloc(img_in.w * img_in.h * sizeof(float));
    img_out.s = (float *)malloc(img_in.w * img_in.h * sizeof(float));
    img_out.l = (unsigned char *)malloc(img_in.w * img_in.h * sizeof(unsigned char));
    
    for(i = 0; i < img_in.w*img_in.h; i ++){
        
        float var_r = ( (float)img_in.img_r[i]/255 );//Convert RGB to [0,1]
        float var_g = ( (float)img_in.img_g[i]/255 );
        float var_b = ( (float)img_in.img_b[i]/255 );
        float var_min = (var_r < var_g) ? var_r : var_g;
        var_min = (var_min < var_b) ? var_min : var_b;   //min. value of RGB
        float var_max = (var_r > var_g) ? var_r : var_g;
        var_max = (var_max > var_b) ? var_max : var_b;   //max. value of RGB
        float del_max = var_max - var_min;               //Delta RGB value
        
        L = ( var_max + var_min ) / 2;
        if ( del_max == 0 )//This is a gray, no chroma...
        {
            H = 0;         
            S = 0;    
        }
        else                                    //Chromatic data...
        {
            if ( L < 0.5 )
                S = del_max/(var_max+var_min);
            else
                S = del_max/(2-var_max-var_min );

            float del_r = (((var_max-var_r)/6)+(del_max/2))/del_max;
            float del_g = (((var_max-var_g)/6)+(del_max/2))/del_max;
            float del_b = (((var_max-var_b)/6)+(del_max/2))/del_max;
            if( var_r == var_max ){
                H = del_b - del_g;
            }
            else{       
                if( var_g == var_max ){
                    H = (1.0/3.0) + del_r - del_b;
                }
                else{
                        H = (2.0/3.0) + del_g - del_r;
                }   
            }
            
        }
        
        if ( H < 0 )
            H += 1;
        if ( H > 1 )
            H -= 1;

        img_out.h[i] = H;
        img_out.s[i] = S;
        img_out.l[i] = (unsigned char)(L*255);
    }
    
    return img_out;
}

float Hue_2_RGB( float v1, float v2, float vH )             //Function Hue_2_RGB
{
    if ( vH < 0 ) vH += 1;
    if ( vH > 1 ) vH -= 1;
    if ( ( 6 * vH ) < 1 ) return ( v1 + ( v2 - v1 ) * 6 * vH );
    if ( ( 2 * vH ) < 1 ) return ( v2 );
    if ( ( 3 * vH ) < 2 ) return ( v1 + ( v2 - v1 ) * ( ( 2.0f/3.0f ) - vH ) * 6 );
    return ( v1 );
}

//Convert HSL to RGB, assume H, S in [0.0, 1.0] and L in [0, 255]
//Output R,G,B in [0, 255]
PPM_IMG hsl2rgb(HSL_IMG img_in)
{
    int i;
    PPM_IMG result;
    
    result.w = img_in.width;
    result.h = img_in.height;
    result.img_r = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_g = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_b = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    for(i = 0; i < img_in.width*img_in.height; i ++){
        float H = img_in.h[i];
        float S = img_in.s[i];
        float L = img_in.l[i]/255.0f;
        float var_1, var_2;
        
        unsigned char r,g,b;
        
        if ( S == 0 )
        {
            r = L * 255;
            g = L * 255;
            b = L * 255;
        }
        else
        {
            
            if ( L < 0.5 )
                var_2 = L * ( 1 + S );
            else
                var_2 = ( L + S ) - ( S * L );

            var_1 = 2 * L - var_2;
            r = 255 * Hue_2_RGB( var_1, var_2, H + (1.0f/3.0f) );
            g = 255 * Hue_2_RGB( var_1, var_2, H );
            b = 255 * Hue_2_RGB( var_1, var_2, H - (1.0f/3.0f) );
        }
        result.img_r[i] = r;
        result.img_g[i] = g;
        result.img_b[i] = b;
    }

    return result;
}

//Convert RGB to YUV, all components in [0, 255]
YUV_IMG rgb2yuv(PPM_IMG img_in)
{
    YUV_IMG img_out;
    int i;//, j;
    unsigned char r, g, b;
    unsigned char y, cb, cr;
    
    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_y = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_u = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_v = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    for(i = 0; i < img_out.w*img_out.h; i ++){
        r = img_in.img_r[i];
        g = img_in.img_g[i];
        b = img_in.img_b[i];
        
        y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
        cb = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
        cr = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
        
        img_out.img_y[i] = y;
        img_out.img_u[i] = cb;
        img_out.img_v[i] = cr;
    }
    
    return img_out;
}

unsigned char clip_rgb(int x)
{
    if(x > 255)
        return 255;
    if(x < 0)
        return 0;

    return (unsigned char)x;
}

//Convert YUV to RGB, all components in [0, 255]
PPM_IMG yuv2rgb(YUV_IMG img_in)
{
    PPM_IMG img_out;
    int i;
    int  rt,gt,bt;
    int y, cb, cr;
    
    
    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_r = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_g = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_b = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    for(i = 0; i < img_out.w*img_out.h; i ++){
        y  = (int)img_in.img_y[i];
        cb = (int)img_in.img_u[i] - 128;
        cr = (int)img_in.img_v[i] - 128;
        
        rt  = (int)( y + 1.402*cr);
        gt  = (int)( y - 0.344*cb - 0.714*cr);
        bt  = (int)( y + 1.772*cb);

        img_out.img_r[i] = clip_rgb(rt);
        img_out.img_g[i] = clip_rgb(gt);
        img_out.img_b[i] = clip_rgb(bt);
    }
    
    return img_out;
}


PGM_IMG gpu_contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int hist[256];
	int img_size = 0;
	int grey_count = 0;
    
	unsigned char * cuda_img_in = 0;
	unsigned char * cuda_img_out = 0;
	int * cuda_img_size = 0;
	//int * cuda_grey_count = 0;
	int * cuda_hist = 0;
	

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
	hipMalloc(&cuda_img_in, sizeof(unsigned char) * result.w * result.h);
	hipMalloc(&cuda_img_out, sizeof(unsigned char) * result.w * result.h);
	hipMalloc(&cuda_img_size, sizeof(int));
	//hipMalloc(&cuda_grey_count, sizeof(int));
	hipMalloc(&cuda_hist, sizeof(int) * 256);

	img_size = img_in.h * img_in.w;
	grey_count = 256;

	hipMemcpy(cuda_img_in, img_in.img, sizeof(unsigned char) * result.w * result.h, hipMemcpyHostToDevice);
	hipMemcpy(cuda_img_size, &img_size, sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(cuda_grey_count, &grey_count, sizeof(int), hipMemcpyHostToDevice);

	/*for (int i = 0; i < 256; i++) {
		hist[i] = 0;
	}*/
	hipMemset(cuda_hist, 0, sizeof(int) * 256);
	//hipMemcpy(cuda_hist, hist, sizeof(int) * 256, hipMemcpyHostToDevice);
	int block_count = (int)ceil((float)img_size / MAXTHREADS);
	/*if (img_size >= grey_count) {
		
		gpu_histogram<<< block_count, MAXTHREADS >>>(cuda_hist, cuda_img_in, cuda_img_size, cuda_grey_count);
	} else {
		gpu_histogram<<< 1, MAXTHREADS >>>(cuda_hist, cuda_img_in, cuda_img_size, cuda_grey_count);
	}*/
	gpu_histogram<<< block_count, MAXTHREADS >>>(cuda_hist, cuda_img_in, cuda_img_size);
	//hipMemset(cuda_hist, 0, sizeof(int) * 256);
	hipMemcpy(hist, cuda_hist, sizeof(int) * 256, hipMemcpyDeviceToHost);
	/*for ( int i = 0; i < img_size; i ++){
		hist[img_in.img[i]] ++;
	}*/
	
	gpu_histogram_equalization(result.img, img_in.img, hist, img_size, grey_count);

	//hipMemcpy(result.img, cuda_img_out, sizeof(unsigned char) * result.w * result.h, hipMemcpyDeviceToHost);
    //histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

	hipFree(cuda_img_in);
	hipFree(cuda_img_out);
	hipFree(cuda_img_size);
	//hipFree(cuda_grey_count);
	hipFree(cuda_hist);
	//free(hist);

	return result;
}

PPM_IMG gpu_contrast_enhancement_c_hsl(PPM_IMG img_in)
{
    HSL_IMG hsl_med;
    PPM_IMG result;
    
    unsigned char * l_equ;
    int hist[256];

    hsl_med = rgb2hsl(img_in);
    l_equ = (unsigned char *)malloc(hsl_med.height*hsl_med.width*sizeof(unsigned char));

    histogram(hist, hsl_med.l, hsl_med.height * hsl_med.width, 256);

    gpu_histogram_equalization(l_equ, hsl_med.l,hist,hsl_med.width*hsl_med.height, 256);
    
    free(hsl_med.l);
    hsl_med.l = l_equ;

    result = hsl2rgb(hsl_med);

    free(hsl_med.h);
    free(hsl_med.s);
    free(hsl_med.l);
    return result;
}


PPM_IMG gpu_contrast_enhancement_c_yuv(PPM_IMG img_in)
{
    //host vars
    PPM_IMG result;
    unsigned char * yuv_med_img_y;
    unsigned char * yuv_med_img_u;
    unsigned char * yuv_med_img_v;
    unsigned char * y_equ;
    int hist[256];
    int image_size = img_in.w * img_in.h;

    //device vars
    //PPM_IMG* gpu_result;
    unsigned char * gpu_result_img_r = 0;
    unsigned char * gpu_result_img_g = 0;
    unsigned char * gpu_result_img_b = 0;
    //PPM_IMG* gpu_img_in;
    unsigned char * gpu_img_in_img_r = 0;
    unsigned char * gpu_img_in_img_g = 0;
    unsigned char * gpu_img_in_img_b = 0;
    //YUV_IMG* gpu_yuv_med;
    unsigned char * gpu_yuv_med_img_y = 0;
    unsigned char * gpu_yuv_med_img_u = 0;
    unsigned char * gpu_yuv_med_img_v = 0;
    int * gpu_image_size;
    int * gpu_hist = 0;

    int block_count = (int)ceil((float)image_size / MAXTHREADS);

    //setup host vars
    yuv_med_img_y = (unsigned char *)malloc(sizeof(unsigned char)*image_size);
    yuv_med_img_u = (unsigned char *)malloc(sizeof(unsigned char)*image_size);
    yuv_med_img_v = (unsigned char *)malloc(sizeof(unsigned char)*image_size);

    //allocate variables for gpu_rgb2yuv
    //Pointers to device memory inside the structure still need to be allocated and freed individually.

    HANDLE_ERROR( hipMalloc(&gpu_img_in_img_r, sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMemcpy(gpu_img_in_img_r, img_in.img_r, sizeof(unsigned char) * image_size, hipMemcpyHostToDevice) );
    
    HANDLE_ERROR( hipMalloc(&gpu_img_in_img_g, sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMemcpy(gpu_img_in_img_g, img_in.img_g, sizeof(unsigned char) * image_size, hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMalloc(&gpu_img_in_img_b, sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMemcpy(gpu_img_in_img_b, img_in.img_b, sizeof(unsigned char) * image_size, hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMalloc(&gpu_yuv_med_img_y, sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMalloc(&gpu_yuv_med_img_u, sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMalloc(&gpu_yuv_med_img_v, sizeof(unsigned char) * image_size) );


    HANDLE_ERROR( hipMalloc(&gpu_image_size, sizeof(int)) );
    HANDLE_ERROR( hipMemcpy(gpu_image_size, &(image_size), sizeof(int), hipMemcpyHostToDevice) );


    //convert to yuv
    gpu_rgb2yuv<<< block_count, MAXTHREADS >>>(gpu_image_size, gpu_img_in_img_r, gpu_img_in_img_g, gpu_img_in_img_b,
                                                gpu_yuv_med_img_y, gpu_yuv_med_img_u, gpu_yuv_med_img_v);

    //********************* done converting up to here

    //copy back to host
    HANDLE_ERROR( hipMemcpy(yuv_med_img_u, gpu_yuv_med_img_u, sizeof(unsigned char) * image_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(yuv_med_img_v, gpu_yuv_med_img_v, sizeof(unsigned char) * image_size, hipMemcpyDeviceToHost) );
    //free used data
    HANDLE_ERROR( hipFree(gpu_img_in_img_r) );
    HANDLE_ERROR( hipFree(gpu_img_in_img_g) );
    HANDLE_ERROR( hipFree(gpu_img_in_img_b) );

    y_equ = (unsigned char *)malloc(image_size*sizeof(unsigned char));

    //setup hist for gpu
    hipMalloc(&gpu_hist, sizeof(int) * 256);
    hipMemset(gpu_hist, 0, sizeof(int) * 256);
    
    gpu_histogram<<< block_count, MAXTHREADS >>>(gpu_hist, gpu_yuv_med_img_y, gpu_image_size);

    HANDLE_ERROR( hipMemcpy(yuv_med_img_y, gpu_yuv_med_img_y, sizeof(unsigned char) * image_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(hist, gpu_hist, sizeof(int) * 256, hipMemcpyDeviceToHost) );
    
    gpu_histogram_equalization(y_equ, yuv_med_img_y, hist, image_size, 256);

    free(yuv_med_img_y);
    yuv_med_img_y = (unsigned char *)malloc(sizeof(unsigned char)*image_size);
    yuv_med_img_y = y_equ;

    //start allocate for converting back to rgb
    HANDLE_ERROR( hipMalloc(&(gpu_result_img_r), sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMalloc(&(gpu_result_img_g), sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMalloc(&(gpu_result_img_b), sizeof(unsigned char) * image_size) );
    HANDLE_ERROR( hipMemcpy(gpu_yuv_med_img_y, yuv_med_img_y, sizeof(unsigned char) * image_size, hipMemcpyHostToDevice) );
    
    //convert back to rgb
    gpu_yuv2rgb<<< block_count, MAXTHREADS >>>(gpu_image_size, gpu_yuv_med_img_y, gpu_yuv_med_img_u, gpu_yuv_med_img_v, 
                                                gpu_result_img_r, gpu_result_img_g, gpu_result_img_b);

    result.img_r = (unsigned char *)malloc(sizeof(unsigned char)*image_size);
    result.img_g = (unsigned char *)malloc(sizeof(unsigned char)*image_size);
    result.img_b = (unsigned char *)malloc(sizeof(unsigned char)*image_size);

    //copy back to host
    HANDLE_ERROR( hipMemcpy(result.img_r, gpu_result_img_r, sizeof(unsigned char)*image_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(result.img_g, gpu_result_img_g, sizeof(unsigned char)*image_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(result.img_b, gpu_result_img_b, sizeof(unsigned char)*image_size, hipMemcpyDeviceToHost) );

    result.w = img_in.w;
    result.h = img_in.h;

    free(yuv_med_img_y); //by freeing this you are freeing y_equ as well
    free(yuv_med_img_u);
    free(yuv_med_img_v);
    HANDLE_ERROR( hipFree(gpu_result_img_r) );
    HANDLE_ERROR( hipFree(gpu_result_img_g) );
    HANDLE_ERROR( hipFree(gpu_result_img_b) );
    HANDLE_ERROR( hipFree(gpu_yuv_med_img_y) );
    HANDLE_ERROR( hipFree(gpu_yuv_med_img_u) );
    HANDLE_ERROR( hipFree(gpu_yuv_med_img_v) );

    return result;
}




//Convert RGB to YUV, all components in [0, 255]
__global__ void gpu_rgb2yuv(int* image_size, unsigned char* img_in_r, unsigned char* img_in_g, unsigned char* img_in_b,
                            unsigned char* img_out_y, unsigned char* img_out_u, unsigned char* img_out_v)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned char r, g, b;
    unsigned char y, cb, cr;

    if(i < *image_size) {
        r = img_in_r[i];
        g = img_in_g[i];
        b = img_in_b[i];
        
        y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
        cb = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
        cr = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
        
        img_out_y[i] = y;
        img_out_u[i] = cb;
        img_out_v[i] = cr;
    }
}

//Convert YUV to RGB, all components in [0, 255]
__global__ void gpu_yuv2rgb(int* image_size, unsigned char* img_in_y, unsigned char* img_in_u, unsigned char* img_in_v,
                            unsigned char* img_out_r, unsigned char* img_out_g, unsigned char* img_out_b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int  rt,gt,bt;
    int y, cb, cr;

    if(i < *image_size) {
        y  = (int)img_in_y[i];
        cb = (int)img_in_u[i] - 128;
        cr = (int)img_in_v[i] - 128;
        
        rt  = (int)( y + 1.402*cr);
        gt  = (int)( y - 0.344*cb - 0.714*cr);
        bt  = (int)( y + 1.772*cb);

        // img_out_r[i] = rt;
        // img_out_g[i] = gt;
        // img_out_b[i] = bt;

        // img_out_r[i] = rt > 255 ? 255 : rt < 0 ? 0 : (unsigned char) rt;
        // img_out_g[i] = gt > 255 ? 255 : gt < 0 ? 0 : (unsigned char) gt;
        // img_out_b[i] = bt > 255 ? 255 : bt < 0 ? 0 : (unsigned char) bt;

        img_out_r[i] = (rt&(~0xFF)) ? (unsigned char)(-rt)>>31 : (unsigned char) rt;
        img_out_g[i] = (gt&(~0xFF)) ? (unsigned char)(-gt)>>31 : (unsigned char) gt;
        img_out_b[i] = (bt&(~0xFF)) ? (unsigned char)(-bt)>>31 : (unsigned char) bt;
    }
}