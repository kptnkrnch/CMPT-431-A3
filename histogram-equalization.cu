#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.cuh"


void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    int i;
    for ( i = 0; i < nbr_bin; i ++){
        hist_out[i] = 0;
    }

    for ( i = 0; i < img_size; i ++){
        hist_out[img_in[i]] ++;
    }
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin){
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++];
    }
    d = img_size - min;
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }
        
        
    }
    
    /* Get the result image */
    for(i = 0; i < img_size; i ++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
        
    }
}

__global__ void gpu_histogram(int * hist_out, unsigned char * img_in, int * img_size, int * nbr_bin){
	int id = threadIdx.x;
    int i;
    //for ( i = 0; i < nbr_bin; i ++){
	if (id < *nbr_bin) {
        hist_out[id] = 0;
	}
    //}

    //for ( i = 0; i < img_size; i ++){
	if (id < *img_size) {
        hist_out[img_in[id]] ++;
	}
    //}
}

void gpu_histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin){
    int *lut = (int *)malloc(sizeof(int)* nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++];
    }
    d = img_size - min;
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }
        
        
    }
    
    /* Get the result image */
    for(i = 0; i < img_size; i ++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
        
    }
}

void gpu_histogram_equalization_calc(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int * lut, int * img_size, int * nbr_bin, int * min){
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++];
    }
    d = img_size - min;
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }
        
        
    }
    
    /* Get the result image */
    for(i = 0; i < img_size; i ++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
        
    }
}